#include "hip/hip_runtime.h"
#include "utility.h"
#include "subckt.h"
#include "defines.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
ARRAY2D<int32_t> gpuAllocateBlockResults(size_t height) {
	int32_t* tgt = NULL;
	hipMalloc(&tgt, sizeof(int)*(height));
	hipMemset(tgt, -1, sizeof(int)*height);
	return ARRAY2D<int32_t>(tgt, 1, height, sizeof(int32_t)*height);
}
void checkCudaError(const char* file, int line) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) { DPRINT("Error %s : before %s:%d\n", hipGetErrorString(err),file,line);}
}
void selectGPU() {
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	hipDeviceProp_t properties;
	if (num_devices > 1) {
		unsigned int max_multiprocessors = 0, max_device = 0;
		for (device = 0; device < num_devices; device++) {
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.totalGlobalMem) {
				max_multiprocessors = properties.totalGlobalMem;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
		hipGetDeviceProperties(&properties, max_device);
		DPRINT("Selected %s as GPU.\n", properties.name);
	}
}

int gpuCalculateSimulPatterns(int lines, int patterns) {
	// get free memory
	size_t free_mem, total_mem;
	int allowed_patterns;
	hipMemGetInfo(&free_mem, &total_mem);
	// added a buffer 	
	allowed_patterns = (free_mem + (lines*sizeof(int))) / (lines*(sizeof(uint32_t)*2.5) + sizeof(uint8_t)*1.5);
	return min(patterns, allowed_patterns -(allowed_patterns % 32));
}
std::string gpuMemCheck(){
	size_t free_mem, total_mem;
	std::stringstream temp;
	hipMemGetInfo(&free_mem, &total_mem);
	temp << free_mem;
	return temp.str();
}

GPU_SCKT_BATCH gpuLoadSubCkts(std::vector<SubCkt>::iterator start, std::vector<SubCkt>::iterator end) {
	int dist = std::distance(start,end);
	int **h_sckt_path = (int**)malloc(sizeof(int*)*dist);
	size_t *h_sckt_sizes = (size_t*)malloc(sizeof(size_t)*dist);
	int **sckt_path;
	size_t *sckt_sizes;
	checkCudaError(__FILE__,__LINE__);
	for (int i = 0; (start+i) < end; i++) {
		h_sckt_path[i] = (start+i)->gpu();
		h_sckt_sizes = (start+i)->size();
	}
	hipMalloc(&sckt_path, sizeof(int*)*dist);
	hipMalloc(&sckt_sizes, sizeof(size_t)*dist);
	checkCudaError(__FILE__,__LINE__);
	hipMemcpy(sckt_path, h_sckt_path, sizeof(int*)*dist,hipMemcpyHostToDevice);
	hipMemcpy(sckt_sizes, h_sckt_sizes, sizeof(size_t)*dist,hipMemcpyHostToDevice);

	free(h_sckt_path);
	free(h_sckt_sizes);
	return GPU_SCKT_BATCH(sckt_path, sckt_sizes);
}
