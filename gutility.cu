#include "hip/hip_runtime.h"
#include "utility.h"
#include "subckt.h"
#include "defines.h"
#include <hip/hip_runtime.h>
#include <stdint.h>
extern int verbose_flag;
ARRAY2D<int32_t> gpuAllocateBlockResults(size_t height) {
	int32_t* tgt = NULL;
	hipMalloc(&tgt, sizeof(int)*(height));
	hipMemset(tgt, -1, sizeof(int)*height);
	return ARRAY2D<int32_t>(tgt, 1, height, sizeof(int32_t)*height);
}
void checkCudaError(const char* file, int line) {
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) { DPRINT("Error %s : before %s:%d\n", hipGetErrorString(err),file,line);}
}
uint8_t selectGPU() {
	int num_devices, device;
	hipGetDeviceCount(&num_devices);
	hipDeviceProp_t properties;
	if (num_devices > 1) {
		unsigned int max_multiprocessors = 0, max_device = 0;
		for (device = 0; device < num_devices; device++) {
			hipGetDeviceProperties(&properties, device);
			if (max_multiprocessors < properties.totalGlobalMem) {
				max_multiprocessors = properties.totalGlobalMem;
				max_device = device;
			}
		}
		hipSetDevice(max_device);
		hipGetDeviceProperties(&properties, max_device);
		hipDeviceSetCacheConfig(hipFuncCachePreferL1);

		DPRINT("Selected %s as GPU.\n", properties.name);
		return max_device;
	}
	return 0;
}
size_t gpuCheckMemory() {
	size_t freeMem = 0;
	size_t totalMem = 0;
	hipMemGetInfo(&freeMem, &totalMem);  
	DPRINT("Memory avaliable: Free: %lu, Total: %lu\n",freeMem, totalMem); 
	return freeMem;

}
#define DIV_UP(x, y) ( (y) * ( ((x)+(y)-1) / (y) ) )
int gpuCalculateSimulPatterns(unsigned int lines, unsigned int patterns, uint8_t deviceID) {
	// get free memory
	hipDeviceProp_t gprop; 
	hipGetDeviceProperties(&gprop, deviceID);
	DPRINT("Calculating for %u lines, %u patterns\n", lines, patterns);
	size_t free_mem;
	free_mem = gpuCheckMemory();
	int allowed_patterns;
	// added a buffer
	allowed_patterns = (free_mem - sizeof(int2)*lines ) / (lines*(sizeof(int32_t)+sizeof(uint8_t)));
	allowed_patterns *=.8;
	DPRINT("Allowed patterns: %d, ", allowed_patterns);
	//allowed_patterns = (free_mem + (lines*sizeof(uint32_t))) / (lines*(sizeof(uint32_t)*4) + sizeof(uint8_t)*1.5);
	while (DIV_UP(allowed_patterns*sizeof(int32_t), gprop.textureAlignment) > allowed_patterns*sizeof(int32_t)) {
		allowed_patterns-=1;
	}
	allowed_patterns /= 32;
	allowed_patterns *= 32;
	DPRINT("%d corrected for 32\n", allowed_patterns);
	return min(patterns, allowed_patterns);
}
std::string gpuMemCheck(){
	size_t free_mem, total_mem;
	std::stringstream temp;
	hipMemGetInfo(&free_mem, &total_mem);
	temp << free_mem;
	return temp.str();
}
void resetGPU() { hipDeviceReset();}
