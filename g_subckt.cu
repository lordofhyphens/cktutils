#include "subckt.h"
#ifndef CPU
void SubCkt::copy() {
	int *test = flat();
	hipMalloc(&_gpu, sizeof(int)*(size()+1));
	hipMemcpy(_gpu, test, sizeof(int)*(size()+1), hipMemcpyHostToDevice);
	delete test;
}

void SubCkt::clear() {
	if (_gpu != NULL) 
		hipFree(_gpu);
	_gpu = NULL;
}
SubCkt::~SubCkt() {
	if (_gpu != NULL) 
		hipFree(_gpu);
	delete _levels;
	delete _subckt;
}

#endif // don't compile this is CPU is defined
