#include "subckt.h"
#ifndef CPU

void SubCkt::copy() {
	hipError_t resp;
	int32_t* flat = new int32_t[_subckt->size()+1];
	for (unsigned int i = 0; i < _subckt->size(); i++)
		flat[i] = _subckt->at(i);
	flat[_subckt->size()] = -1;

	resp = hipMalloc(&_gpu, sizeof(int)*(size()+1));
	if (resp != hipSuccess) { 
		DPRINT("Error allocating memory for GPU copy of subckt\n");
	}
	resp = hipMemcpy(_gpu, flat, sizeof(int)*(size()+1), hipMemcpyHostToDevice);
	if (resp != hipSuccess) { 
		DPRINT("Error copying GPU copy of subckt to GPU\n");
	}
	delete [] flat;
}

void SubCkt::clear() {
	if (_gpu != NULL) 
		hipFree(_gpu);
	_gpu = NULL;
}
SubCkt::~SubCkt() {
	if (_gpu != NULL) 
		hipFree(_gpu);
	delete _levels;
	delete _subckt;
}

#endif // don't compile this is CPU is defined
